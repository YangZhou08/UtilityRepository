#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <chrono>

// Initialize random values for matrices
void initializeMatrix(std::vector<float>& mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Function to perform cuBLAS matrix multiplication
void runCuBLAS(int m, int n, int k) {
    // Host matrices
    std::vector<float> h_A(m * k);
    std::vector<float> h_B(k * n);
    std::vector<float> h_C(m * n);

    // Initialize matrices with random values
    initializeMatrix(h_A, m, k);
    initializeMatrix(h_B, k, n);

    // Device matrices
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A.data(), m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, m * n * sizeof(float));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform matrix multiplication
    const float alpha = 1.0f, beta = 0.0f;

    auto start = std::chrono::high_resolution_clock::now();

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                m, n, k,             // Matrix dimensions
                &alpha,              // Alpha
                d_A, m,              // Matrix A and leading dimension
                d_B, k,              // Matrix B and leading dimension
                &beta,               // Beta
                d_C, m);             // Matrix C and leading dimension

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Matrix multiplication (m=" << m << ", n=" << n << ", k=" << k
              << ") took " << elapsed.count() << " seconds." << std::endl;

    // Copy result back to host
    hipMemcpy(h_C.data(), d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}

int main() {
    // Example: Multiply two 1024 x 1024 matrices
    int m = 1024, n = 1024, k = 1024;

    std::cout << "Starting cuBLAS matrix multiplication..." << std::endl;
    runCuBLAS(m, n, k);

    return 0;
} 