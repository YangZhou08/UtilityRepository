#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h> // For __half type
#include <cusparseLt.h>

// Define matrix dimensions and parameters
#define NUM_A_ROWS 128 // Number of rows in matrix A
#define NUM_A_COLS 128 // Number of columns in matrix A
#define NUM_B_ROWS 128 // Number of rows in matrix B
#define NUM_B_COLS 128 // Number of columns in matrix B
#define NUM_C_ROWS NUM_A_ROWS // Number of rows in matrix C
#define NUM_C_COLS NUM_B_COLS // Number of columns in matrix C
#define LDA NUM_A_COLS        // Leading dimension of matrix A
#define LDB NUM_B_COLS        // Leading dimension of matrix B
#define LDC NUM_C_COLS        // Leading dimension of matrix C
#define ALIGNMENT 16          // Memory alignment

int main() {
    // Scalars for multiplication
    float alpha = 1.0f;
    float beta = 0.0f;

    // CUDA stream
    hipStream_t stream = nullptr;

    // Create cuSPARSELt handle
    cusparseLtHandle_t handle;
    cusparseLtInit(&handle);

    // Define matrix data types
    hipDataType type = HIP_R_16F; // Half precision (__half)
    hipsparseOrder_t order = HIPSPARSE_ORDER_ROW; // Row-major order
    cusparseComputeType compute_type = CUSPARSE_COMPUTE_32F; // Compute in single precision

    // Allocate memory for matrices on the device
    __half *dA, *dB, *dC, *dD;
    hipMalloc(&dA, NUM_A_ROWS * NUM_A_COLS * sizeof(__half));
    hipMalloc(&dB, NUM_B_ROWS * NUM_B_COLS * sizeof(__half));
    hipMalloc(&dC, NUM_C_ROWS * NUM_C_COLS * sizeof(__half));
    hipMalloc(&dD, NUM_C_ROWS * NUM_C_COLS * sizeof(__half));

    // Fill matrices with random data (for simplicity, using memset here)
    hipMemset(dA, 1, NUM_A_ROWS * NUM_A_COLS * sizeof(__half));
    hipMemset(dB, 1, NUM_B_ROWS * NUM_B_COLS * sizeof(__half));
    hipMemset(dC, 0, NUM_C_ROWS * NUM_C_COLS * sizeof(__half));

    // Initialize matrix descriptors
    cusparseLtMatDescriptor_t matA, matB, matC;
    cusparseLtStructuredDescriptorInit(&handle, &matA, NUM_A_ROWS, NUM_A_COLS, LDA, ALIGNMENT, type, order, CUSPARSELT_SPARSITY_50_PERCENT);
    cusparseLtDenseDescriptorInit(&handle, &matB, NUM_B_ROWS, NUM_B_COLS, LDB, ALIGNMENT, type, order);
    cusparseLtDenseDescriptorInit(&handle, &matC, NUM_C_ROWS, NUM_C_COLS, LDC, ALIGNMENT, type, order);

    // Initialize matmul descriptor and plan
    cusparseLtMatmulDescriptor_t matmul;
    cusparseLtMatmulAlgSelection_t alg_sel;
    cusparseLtMatmulPlan_t plan;
    cusparseLtMatmulDescriptorInit(&handle, &matmul, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &matA, &matB, &matC, &matC, compute_type);
    cusparseLtMatmulAlgSelectionInit(&handle, &alg_sel, &matmul, CUSPARSELT_MATMUL_ALG_DEFAULT);
    cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel);

    // Prune matrix A
    cusparseLtSpMMAPrune(&handle, &matmul, dA, dA, CUSPARSELT_PRUNE_SPMMA_TILE, stream);

    // Check pruning correctness
    int *d_valid;
    hipMalloc((void **)&d_valid, sizeof(int));
    cusparseLtSpMMAPruneCheck(&handle, &matmul, dA, d_valid, stream);

    int is_valid;
    hipMemcpyAsync(&is_valid, d_valid, sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    if (is_valid != 0) {
        std::cerr << "Pruned matrix A is invalid!" << std::endl;
        return EXIT_FAILURE;
    }

    // Compress matrix A
    size_t compressed_size;
    cusparseLtSpMMACompressedSize(&handle, &plan, &compressed_size);
    void *dA_compressed;
    hipMalloc(&dA_compressed, compressed_size);
    cusparseLtSpMMACompress(&handle, &plan, dA, dA_compressed, stream);

    // Allocate workspace
    size_t workspace_size;
    cusparseLtMatmulGetWorkspace(&handle, &plan, &workspace_size);
    void *d_workspace = nullptr;
    if (workspace_size > 0) {
        hipMalloc(&d_workspace, workspace_size);
    }

    // Perform matrix multiplication
    cusparseLtMatmul(&handle, &plan, &alpha, dA_compressed, dB, &beta, dC, dD, d_workspace, &stream, 0);

    // Cleanup
    cusparseLtMatDescriptorDestroy(&matA);
    cusparseLtMatDescriptorDestroy(&matB);
    cusparseLtMatDescriptorDestroy(&matC);
    cusparseLtMatmulPlanDestroy(&plan);
    cusparseLtDestroy(&handle);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dD);
    hipFree(dA_compressed);
    hipFree(d_workspace);
    hipFree(d_valid);

    std::cout << "Matrix multiplication completed successfully!" << std::endl;
    return EXIT_SUCCESS;
} 
