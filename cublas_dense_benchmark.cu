#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <chrono>

// Initialize random values for matrices
void initializeMatrix(std::vector<float>& mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Function to perform cuBLAS matrix multiplication
void runDenseMatmul(int m, int n, int k) {
    // Host matrices
    std::vector<float> h_A(m * k);
    std::vector<float> h_B(k * n);
    std::vector<float> h_C(m * n);

    // Initialize matrices with random values
    initializeMatrix(h_A, m, k);
    initializeMatrix(h_B, k, n);

    // Device matrices
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float)); 

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // Copy host matrices to device
    hipMemcpy(d_A, h_A.data(), m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, m * n * sizeof(float)); 

    float elapsed_time_ms = 0.0f; 

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle); 

    int num_iterations = 1000;  

    // Perform matrix multiplication
    const float alpha = 1.0f, beta = 0.0f; 

    for (int i = 0; i < 10; ++i) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    m, n, k,             // Matrix dimensions
                    &alpha,              // Alpha
                    d_A, m,              // Matrix A and leading dimension
                    d_B, k,              // Matrix B and leading dimension
                    &beta,               // Beta
                    d_C, m);             // Matrix C and leading dimension 
    } 

    // Record the start event
    hipEventRecord(start, 0); 

    for (int i = 0; i < num_iterations; ++i) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    m, n, k,             // Matrix dimensions
                    &alpha,              // Alpha
                    d_A, m,              // Matrix A and leading dimension
                    d_B, k,              // Matrix B and leading dimension
                    &beta,               // Beta
                    d_C, m);             // Matrix C and leading dimension 
    } 

    // Record the stop event 
    // hipDeviceSynchronize(); 
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop); 
    // hipEventSynchronize(stop); 

    // Calculate the elapsed time
    hipEventElapsedTime(&elapsed_time_ms, start, stop); 

    // Calculate average runtime
    double avg_time_per_iteration = elapsed_time_ms.count() / num_iterations; 

    std::cout << "Sparse matrix multiplication (m=" << m << ", n=" << n << ", k=" << k
          << ") average runtime over " << num_iterations << " iterations: "
          << avg_time_per_iteration << " seconds." << std::endl; 

    // std::cout << "Dense matrix multiplication (m=" << m << ", n=" << n << ", k=" << k
    //           << ") took " << elapsed.count() << " seconds." << std::endl; 

    // Copy result back to host
    hipMemcpy(h_C.data(), d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle); 

    hipEventDestroy(start);
    hipEventDestroy(stop); 
}

int main() {
    // Example: Multiply two 1024 x 1024 matrices
    int m = 1024, n = 1024, k = 1024;

    std::cout << "Starting cuBLAS dense matrix multiplication benchmark..." << std::endl;
    runDenseMatmul(m, n, k);

    return 0;
} 
