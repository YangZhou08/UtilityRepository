#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cusparseLt.h>
#include <vector>
#include <iostream>
#include <chrono>

// Function to initialize matrices with random values
void initializeMatrix(std::vector<__half>& mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Sparse Matrix Multiplication
void runSparseMatmul(int m, int n, int k) {
    // Host matrices
    std::vector<__half> h_A(m * k); // Dense matrix A
    std::vector<__half> h_B(k * n); // Dense matrix B
    std::vector<__half> h_C(m * n); // Result matrix C

    // Initialize matrices with random values
    initializeMatrix(h_A, m, k);
    initializeMatrix(h_B, k, n);

    // Device matrices
    __half *d_A, *d_B, *d_C, *d_A_compressed;
    hipMalloc(&d_A, m * k * sizeof(__half));
    hipMalloc(&d_B, k * n * sizeof(__half));
    hipMalloc(&d_C, m * n * sizeof(__half));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A.data(), m * k * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), k * n * sizeof(__half), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, m * n * sizeof(__half));

    // cuSPARSELt setup
    cusparseLtHandle_t handle;
    cusparseLtInit(&handle);

    cusparseLtMatDescriptor_t matA, matB, matC;
    cusparseLtMatmulDescriptor_t matmul;
    cusparseLtMatmulAlgSelection_t alg_sel;
    cusparseLtMatmulPlan_t plan;

    hipStream_t stream = nullptr;
    size_t compressed_size, compress_buffer_size;
    void* compress_buffer = nullptr;

    // Descriptors
    cusparseLtStructuredDescriptorInit(&handle, &matA, m, k, k, 16, HIP_R_16F, HIPSPARSE_ORDER_ROW, CUSPARSELT_SPARSITY_50_PERCENT);
    cusparseLtDenseDescriptorInit(&handle, &matB, k, n, n, 16, HIP_R_16F, HIPSPARSE_ORDER_ROW);
    cusparseLtDenseDescriptorInit(&handle, &matC, m, n, n, 16, HIP_R_16F, HIPSPARSE_ORDER_ROW);

    // Matmul descriptor
    cusparseLtMatmulDescriptorInit(&handle, &matmul, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &matA, &matB, &matC, &matC, CUSPARSE_COMPUTE_32F);
    cusparseLtMatmulAlgSelectionInit(&handle, &alg_sel, &matmul, CUSPARSELT_MATMUL_ALG_DEFAULT);
    cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel);

    // Prune and compress
    cusparseLtSpMMAPrune(&handle, &matmul, d_A, d_A, CUSPARSELT_PRUNE_SPMMA_TILE, stream);
    cusparseLtSpMMACompressedSize(&handle, &plan, &compressed_size, &compress_buffer_size);

    hipMalloc(&d_A_compressed, compressed_size);
    hipMalloc(&compress_buffer, compress_buffer_size);
    cusparseLtSpMMACompress(&handle, &plan, &matA, d_A, d_A_compressed, compress_buffer, stream);

    // Workspace
    size_t workspace_size;
    void* d_workspace = nullptr;
    cusparseLtMatmulGetWorkspace(&handle, &plan, &workspace_size);
    if (workspace_size > 0) {
        hipMalloc(&d_workspace, workspace_size);
    } 

    int num_iterations = 1000; 

    // Timer
    auto start = std::chrono::high_resolution_clock::now();

    // Matrix multiplication
    float alpha = 1.0f, beta = 0.0f; 
    for (int i = 0; i < num_iterations; ++i) {
        cusparseLtMatmul(&handle, &plan, &alpha, d_A_compressed, d_B, &beta, d_C, d_C, d_workspace, nullptr, 0); 
        hipDeviceSynchronize();
    } 

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start; 

    // Calculate average runtime
    double avg_time_per_iteration = elapsed.count() / num_iterations; 

    std::cout << "Sparse matrix multiplication (m=" << m << ", n=" << n << ", k=" << k
          << ") average runtime over " << num_iterations << " iterations: "
          << avg_time_per_iteration << " seconds." << std::endl; 

    // std::cout << "Sparse matrix multiplication (m=" << m << ", n=" << n << ", k=" << k
    //           << ") took " << elapsed.count() << " seconds." << std::endl; 

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_A_compressed);
    hipFree(compress_buffer);
    if (workspace_size > 0) {
        hipFree(d_workspace);
    }

    cusparseLtMatDescriptorDestroy(&matA);
    cusparseLtMatDescriptorDestroy(&matB);
    cusparseLtMatDescriptorDestroy(&matC);
    cusparseLtMatmulPlanDestroy(&plan);
    cusparseLtDestroy(&handle);
}

int main() {
    // Example: Multiply two 1024 x 1024 matrices
    int m = 1024, n = 1024, k = 1024;

    std::cout << "Starting cuSPARSELt sparse matrix multiplication benchmark..." << std::endl;
    runSparseMatmul(m, n, k);

    return 0;
} 
