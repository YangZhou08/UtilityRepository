#include <hip/hip_runtime.h>
#include <cusparseLt.h>
#include <vector>
#include <iostream>
#include <chrono>

// Function to initialize matrices with random values
void initializeMatrix(std::vector<float>& mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Sparse Matrix Multiplication
void runSparseMatmul(int m, int n, int k) {
    // Host matrices
    std::vector<float> h_A(m * k); // Dense matrix A
    std::vector<float> h_B(k * n); // Dense matrix B
    std::vector<float> h_C(m * n); // Result matrix C

    // Initialize matrices with random values
    initializeMatrix(h_A, m, k);
    initializeMatrix(h_B, k, n);

    // Device matrices
    float *d_A, *d_B, *d_C, *d_A_compressed;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A.data(), m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, m * n * sizeof(float));

    // cuSPARSELt setup
    cusparseLtHandle_t handle;
    cusparseLtInit(&handle);

    cusparseLtMatDescriptor_t matA, matB, matC;
    cusparseLtMatmulDescriptor_t matmul;
    cusparseLtMatmulAlgSelection_t alg_sel;
    cusparseLtMatmulPlan_t plan;

    hipStream_t stream;
    hipStreamCreate(&stream);
    size_t compressed_size, compress_buffer_size;
    void* compress_buffer = nullptr;

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Descriptors - Changed to HIP_R_32F
    cusparseLtStructuredDescriptorInit(&handle, &matA, m, k, k, 16, HIP_R_32F, HIPSPARSE_ORDER_ROW, CUSPARSELT_SPARSITY_50_PERCENT);
    cusparseLtDenseDescriptorInit(&handle, &matB, k, n, n, 16, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    cusparseLtDenseDescriptorInit(&handle, &matC, m, n, n, 16, HIP_R_32F, HIPSPARSE_ORDER_ROW);

    // Matmul descriptor
    cusparseLtMatmulDescriptorInit(&handle, &matmul, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &matA, &matB, &matC, &matC, CUSPARSE_COMPUTE_32F);
    cusparseLtMatmulAlgSelectionInit(&handle, &alg_sel, &matmul, CUSPARSELT_MATMUL_ALG_DEFAULT);
    cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel);

    float elapsed_time_ms = 0.0f;

    // Prune and compress
    cusparseLtSpMMAPrune(&handle, &matmul, d_A, d_A, CUSPARSELT_PRUNE_SPMMA_TILE, stream);
    cusparseLtSpMMACompressedSize(&handle, &plan, &compressed_size, &compress_buffer_size);

    hipMalloc(&d_A_compressed, compressed_size);
    hipMalloc(&compress_buffer, compress_buffer_size);
    
    hipsparseStatus_t compress_status = cusparseLtSpMMACompress(
        &handle,
        &plan,
        d_A,
        d_A_compressed,
        compress_buffer,
        stream
    );

    // Workspace
    size_t workspace_size;
    void* d_workspace = nullptr;
    cusparseLtMatmulGetWorkspace(&handle, &plan, &workspace_size);
    if (workspace_size > 0) {
        hipMalloc(&d_workspace, workspace_size);
    }
    float alpha = 1.0f, beta = 0.0f;

    int num_iterations = 1000;
    for (int i = 0; i < 10; ++i) {
        cusparseLtMatmul(&handle, &plan, &alpha, d_A_compressed, d_B, &beta, d_C, d_C, d_workspace, &stream, 0);
    }

    hipEventRecord(start, stream);

    for (int i = 0; i < num_iterations; ++i) {
        cusparseLtMatmul(&handle, &plan, &alpha, d_A_compressed, d_B, &beta, d_C, d_C, d_workspace, &stream, 0);
    }

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    std::cout << "Elapsed time: " << elapsed_time_ms << " ms" << std::endl;
    double avg_time_per_iteration = elapsed_time_ms / num_iterations;

    std::cout << "Sparse matrix multiplication (m=" << m << ", n=" << n << ", k=" << k
          << ") average runtime over " << num_iterations << " iterations: "
          << avg_time_per_iteration << " ms" << std::endl;

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_A_compressed);
    hipFree(compress_buffer);
    if (workspace_size > 0) {
        hipFree(d_workspace);
    }

    cusparseLtMatDescriptorDestroy(&matA);
    cusparseLtMatDescriptorDestroy(&matB);
    cusparseLtMatDescriptorDestroy(&matC);
    cusparseLtMatmulPlanDestroy(&plan);
    cusparseLtDestroy(&handle);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);
}

int main() {
    // int m = 1024, n = 1024, k = 1024; 
    // int m = 128, n = 128, k = 128; 
    // int m = 256, n = 256, k = 256; 
    // int m = 512, n = 512, k = 512; 
    int m = 1024, n = 1024, k = 1024; 

    std::cout << "Starting cuSPARSELt sparse matrix multiplication benchmark..." << std::endl;
    runSparseMatmul(m, n, k);

    return 0;
} 
